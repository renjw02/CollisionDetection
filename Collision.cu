#include "hip/hip_runtime.h"
﻿#include <math.h>
#include <stdint.h>
#include "Collision.cuh"
#include "Ball.h"

__device__ float dist_(float x, float y, float z)
{
	return sqrt(x * x + y * y + z * z);
}


// 精确碰撞检测
__device__ bool isCollision(Ball& a, Ball& b)
{
	float dist_x = a.pos.x - b.pos.x;
	float dist_y = a.pos.y - b.pos.y;
	float dist_z = a.pos.z - b.pos.z;
	float dist = dist_(dist_x, dist_y, dist_z);
	if (dist < a.radius + b.radius)
	{
		return true;
	}
	else
	{
		return false;
	}
}

// 碰撞后速度更新
__device__ void updateSpeed(Ball& a, Ball& b)
{
	float dist = 0;
	float dist_x = b.pos.x - a.pos.x;
	float dist_y = b.pos.y - a.pos.y;
	float dist_z = b.pos.z - a.pos.z;
	dist = dist_(dist_x, dist_y, dist_z);

	// 碰撞方向的速度
	float rate_collide_a = (a.speed.x * dist_x + a.speed.y * dist_y + a.speed.z * dist_z) / dist / dist;
	float normal_velocity_a_x = dist_x * rate_collide_a;
	float normal_velocity_a_y = dist_y * rate_collide_a;
	float normal_velocity_a_z = dist_z * rate_collide_a;

	float rate_collide_b = (b.speed.x * dist_x + b.speed.y * dist_y + b.speed.z * dist_z) / dist / dist;
	float normal_velocity_b_x = dist_x * rate_collide_b;
	float normal_velocity_b_y = dist_y * rate_collide_b;
	float normal_velocity_b_z = dist_z * rate_collide_b;

	// 垂直方向的速度
	float tangential_velocity_a_x = a.speed.x - normal_velocity_a_x;
	float tangential_velocity_a_y = a.speed.y - normal_velocity_a_y;
	float tangential_velocity_a_z = a.speed.z - normal_velocity_a_z;

	float tangential_velocity_b_x = b.speed.x - normal_velocity_b_x;
	float tangential_velocity_b_y = b.speed.y - normal_velocity_b_y;
	float tangential_velocity_b_z = b.speed.z - normal_velocity_b_z;

	// 更新速度，需考虑弹性系数和质量
	float normal_velocity_new_a_x = ((1 - a.coefficient) * (normal_velocity_a_x * (a.weight - b.weight) + normal_velocity_b_x * (2 * b.weight))) / (a.weight + b.weight);
	float normal_velocity_new_a_y = ((1 - a.coefficient) * (normal_velocity_a_y * (a.weight - b.weight) + normal_velocity_b_y * (2 * b.weight))) / (a.weight + b.weight);
	float normal_velocity_new_a_z = ((1 - a.coefficient) * (normal_velocity_a_z * (a.weight - b.weight) + normal_velocity_b_z * (2 * b.weight))) / (a.weight + b.weight);

	float normal_velocity_new_b_x = ((1 - b.coefficient) * (normal_velocity_a_x * (2 * a.weight) + normal_velocity_b_x * (b.weight - a.weight))) / (a.weight + b.weight);
	float normal_velocity_new_b_y = ((1 - b.coefficient) * (normal_velocity_a_y * (2 * a.weight) + normal_velocity_b_y * (b.weight - a.weight))) / (a.weight + b.weight);
	float normal_velocity_new_b_z = ((1 - b.coefficient) * (normal_velocity_a_z * (2 * a.weight) + normal_velocity_b_z * (b.weight - a.weight))) / (a.weight + b.weight);


	a.speed.x = normal_velocity_new_a_x + tangential_velocity_a_x;
	a.speed.y = normal_velocity_new_a_y + tangential_velocity_a_y;
	a.speed.z = normal_velocity_new_a_z + tangential_velocity_a_z;

	b.speed.x = normal_velocity_new_b_x + tangential_velocity_b_x;
	b.speed.y = normal_velocity_new_b_y + tangential_velocity_b_y;
	b.speed.z = normal_velocity_new_b_z + tangential_velocity_b_z;
}


__global__ void updateBalls(Ball* balls, float interval, float length, float width, float height, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		// s_t = s_0 + v * t
		balls[i].pos.x = balls[i].pos.x + balls[i].speed.x * interval;
		balls[i].pos.y = balls[i].pos.y + balls[i].speed.y * interval;
		balls[i].pos.z = balls[i].pos.z + balls[i].speed.z * interval;

		// 考虑撞墙
		if (balls[i].pos.x - balls[i].radius < -length)
		{
			balls[i].pos.x = -length + balls[i].radius;
			balls[i].speed.x = -balls[i].speed.x;
		}
		else if (balls[i].pos.x + balls[i].radius > length)
		{
			balls[i].pos.x = length - balls[i].radius;
			balls[i].speed.x = -balls[i].speed.x;
		}
		if (balls[i].pos.z - balls[i].radius < -width)
		{
			balls[i].pos.z = -width + balls[i].radius;
			balls[i].speed.z = -balls[i].speed.z;
		}
		else if (balls[i].pos.z + balls[i].radius > width)
		{
			balls[i].pos.z = width - balls[i].radius;
			balls[i].speed.z = -balls[i].speed.z;
		}
		if (balls[i].pos.y - balls[i].radius < 0)
		{
			balls[i].pos.y = balls[i].radius;
			balls[i].speed.y = -balls[i].speed.y;
		}
		else if (balls[i].pos.y + balls[i].radius > height)
		{
			balls[i].pos.y = height - balls[i].radius;
			balls[i].speed.y = -balls[i].speed.y;
		}
	}
}

__global__ void collision(uint32_t* cells, uint32_t* objects, Ball* balls, int num_balls, int num_cells, uint32_t* indices,
	uint32_t num_indices, unsigned int group_per_thread, float length, float width, float height, 
	float gridSize, int grid_x, int grid_y, int grid_z)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	for (int group_num = 0; group_num < group_per_thread; group_num++)
	{
		// broad phase
		// 确定开始和结束的索引
		int cell_id = index * group_per_thread + group_num;
		if (cell_id >= num_indices)
		{
			break;
		}
		int end = indices[cell_id];
		int start = 0;
		if (cell_id == 0)
		{
			start = 0;
		}
		else
		{
			start = indices[cell_id - 1];
		}

		// home的个数
		int home_num = 0;
		for (int i = start; i < end; i++)
		{
			int type = cells[i] & 1;
			if (type == HOME_CELL)
			{
				home_num++;
			}
			else
			{
				break;
			}
		}

		// 碰撞检测
		for (int i = start; i < start + home_num; i++)
		{
			if (cells[i] == UINT32_MAX) break;
			int ball_i = (objects[i] >> 1) & 65535;

			for (int j = i + 1; j < end; j++)
			{
				if (cells[j] == UINT32_MAX) break;
				int ball_j = (objects[j] >> 1) & 65535;
				
				// narrow phase
				// 直接通过球心距判断
				// 
				// 都是home 
				if (j < start + home_num)
				{
					if (isCollision(balls[ball_i], balls[ball_j]))
					{
						updateSpeed(balls[ball_i], balls[ball_j]);
					}
				}
				else
				{
					int home_i = (cells[i] >> 1) & ((1 << 24) - 1);
					int j_x = (balls[ball_j].pos.x + length) / gridSize;
					int j_y = balls[ball_j].pos.y / gridSize;
					int j_z = (balls[ball_j].pos.z + width) / gridSize;
					int home_j = j_x << 16 | j_y << 8 | j_z;

					// home和phantom
					if (home_i < home_j)
					{
						if (isCollision(balls[ball_i], balls[ball_j]))
						{
							updateSpeed(balls[ball_i], balls[ball_j]);
						}
					}
				}
			}
		}

	}
}


// 根据基数求和重新排序
__global__ void arrange(uint32_t* cells, uint32_t* objects, uint32_t* cells_temp, uint32_t* objects_temp, 
	uint32_t* radix_sums, int n, int shift)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int num_radices = 1 << RADIX_LENGTH;

	if (index != 0) return;

	for (int i = 0; i < n; i++)
	{
		int current_radix_num = (cells[i] >> shift) & (num_radices - 1);
		cells_temp[radix_sums[current_radix_num]] = cells[i];
		objects_temp[radix_sums[current_radix_num]] = objects[i];
		radix_sums[current_radix_num]++;
	}
}

// 获取不同网格的起始索引及不同网格单元的数量
__global__ void getCellIndex(uint32_t* cells, int n, uint32_t* indices, uint32_t* indices_num)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index != 0)
		return;
	indices_num[0] = 0;
	uint32_t previous = UINT32_MAX;
	uint32_t current = UINT32_MAX;
	uint32_t mask = (1 << 24) - 1;
	for (int i = 0; i < n; i++)
	{
		current = mask & (cells[i] >> 1);
		if (previous == UINT32_MAX)
		{
			previous = current;
		}
		if (previous != current)
		{
			indices[indices_num[0]] = i;
			indices_num[0]++;
		}
		previous = current;
	}
	indices[indices_num[0]] = n;
	indices_num[0]++;
}

// 基数求和，存在数组中便于后续排序
__global__ void getRadixSum(uint32_t* cells, uint32_t* radix_sums, int n, int shift)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int num_indices = 1 << RADIX_LENGTH;

	for (int i = index; i < num_indices; i++)
	{
		radix_sums[i] = 0;
	}
	__syncthreads();


	// 每个线程处理对应的小球
	for (int i = index; i < n; i += stride)
	{

		for (int j = 0; j < blockDim.x; j++)
		{
			if (threadIdx.x % blockDim.x == j)
			{
				int current_radix_num = (cells[i] >> shift) & (num_indices - 1);
				radix_sums[current_radix_num]++;
			}
		}

	}
	__syncthreads();

	// 获取前缀和
	int o = 1;
	int a;

	// 归约
	for (int d = num_indices / 2; d; d /= 2)
	{
		__syncthreads();

		if (threadIdx.x < d)
		{
			a = (threadIdx.x * 2 + 1) * o - 1;
			radix_sums[a + o] += radix_sums[a];
		}

		o *= 2;
	}
	if (!threadIdx.x)
	{
		radix_sums[num_indices - 1] = 0;
	}

	// 反向传播
	uint32_t temp;
	for (int d = 1; d < num_indices; d *= 2)
	{
		__syncthreads();
		o /= 2;

		if (threadIdx.x < d)
		{
			a = (threadIdx.x * 2 + 1) * o - 1;
			temp = radix_sums[a];
			radix_sums[a] = radix_sums[a + o];
			radix_sums[a + o] += temp;
		}
	}

	__syncthreads();
}

// 初始化球体在空间网格中的位置和信息
__global__ void init(uint32_t* cells, uint32_t* objects, Ball* balls, float length, float width, float height, float gridSize, int grid_x, int grid_y,
	int grid_z, int n)
{
	unsigned int count = 0;

	// 遍历所有球体
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
	{
		// 计算所在网格信息并存入cells和objects数组
		int current_cell_id = i * 8;
		int cell_info = 0;
		int object_info = 0;
		int current_count = 0;
		float x = balls[i].pos.x;
		float y = balls[i].pos.y;
		float z = balls[i].pos.z;
		float radius = balls[i].radius;

		int hash_x = (x + length) / gridSize;
		int hash_y = (y) / gridSize;
		int hash_z = (z + width) / gridSize;
		cell_info = hash_x << 17 | hash_y << 9 | hash_z << 1 | HOME_CELL;
		object_info = i << 1 | HOME_OBJECT;
		cells[current_cell_id] = cell_info;
		objects[current_cell_id] = object_info;
		current_cell_id++;
		count++;
		current_count++;

		// 查找记录相邻的单元格
		for (int dx = -1; dx <= 1; dx++)
		{
			for (int dy = -1; dy <= 1; dy++)
			{
				for (int dz = -1; dz <= 1; dz++)
				{
					int new_hash_x = hash_x + dx;
					int new_hash_y = hash_y + dy;
					int new_hash_z = hash_z + dz;

					// 自己
					if (dx == 0 && dy == 0 && dz == 0)
					{
						continue;
					}

					// 越界
					if (new_hash_x < 0 || new_hash_x >= grid_x ||
						new_hash_y < 0 || new_hash_y >= grid_y ||
						new_hash_z < 0 || new_hash_z >= grid_z)
					{
						continue;
					}

					float relative_x = 0;
					float relative_y = 0;
					float relative_z = 0;
					if (dx == 0)
					{
						relative_x = x;
					}
					else if (dx == -1)
					{
						relative_x = hash_x * gridSize - length;
					}
					else
					{
						relative_x = (hash_x + 1) * gridSize - length;
					}

					if (dz == 0)
					{
						relative_z = z;
					}
					else if (dz == -1)
					{
						relative_z = hash_z * gridSize - width;
					}
					else
					{
						relative_z = (hash_z + 1) * gridSize - width;
					}

					if (dy == 0)
					{
						relative_y = y;
					}
					else if (dy == -1)
					{
						relative_y = hash_y * gridSize;
					}
					else
					{
						relative_y = (hash_y + 1) * gridSize;
					}

					relative_x -= x;
					relative_y -= y;
					relative_z -= z;

					float dist = dist_(relative_x, relative_y, relative_z);
					if (dist < radius)
					{
						int cell_info = new_hash_x << 17 | new_hash_y << 9 | new_hash_z << 1 | PHANTOM_CELL;
						int object_info = i << 1 | PHANTOM_OBJECT;
						cells[current_cell_id] = cell_info;
						objects[current_cell_id] = object_info;
						current_cell_id++;
						count++;
						current_count++;
					}
				}
			}
		}

		// 每个球体的信息应该占据连续的8个位置，空余的要补齐
		while (current_count < 8)
		{

			cells[current_cell_id] = UINT32_MAX;
			objects[current_cell_id] = i << 2;
			current_cell_id++;
			current_count++;
		}

	}

}


void collisionDetection(Ball* balls, float refreshInterval, float length, float width, float height,
	float gridSize, int grid_x, int grid_y, int grid_z, int n)
{

	// GPU上的小球数组
	Ball* g_balls;
	unsigned int nBytes = n * sizeof(Ball);
	hipMalloc((void**)&g_balls, nBytes);

	unsigned int num_blocks = 128;
	unsigned int threads_per_block = 512;
	unsigned int object_size = (n - 1) / threads_per_block + 1;
	if (object_size < num_blocks) {
		num_blocks = object_size;
	}

	// 将cpu数据复制到gpu上
	hipMemcpy((void*)g_balls, (void*)balls, nBytes, hipMemcpyHostToDevice);

	// 更新状态
	updateBalls <<< num_blocks, threads_per_block >>> (g_balls, refreshInterval, length, width, height, n);
	hipDeviceSynchronize();

	// 碰撞检测
	spatialSubdivision(g_balls, length, width, height, gridSize, grid_x, grid_y, grid_z, n, num_blocks, threads_per_block);
	hipDeviceSynchronize();

	// 计算好的结果复制回cpu，更新图形界面
	hipMemcpy((void*)balls, (void*)g_balls, nBytes, hipMemcpyDeviceToHost);

	hipFree(g_balls);
}

void spatialSubdivision(Ball* balls, float length, float width, float height,
	float gridSize, int grid_x, int grid_y, int grid_z, int n, unsigned int num_blocks, unsigned int threads_per_block)
{
	unsigned int cell_size = n * 8 * sizeof(uint32_t);

	int num_radices = 1 << RADIX_LENGTH;
	uint32_t* cells;
	uint32_t* cells_temp;
	uint32_t* objects;
	uint32_t* objects_temp;
	uint32_t* indices;
	uint32_t* indices_num;
	uint32_t* radix_sums;

	hipMalloc((void**)&cells, cell_size);
	hipMalloc((void**)&cells_temp, cell_size);
	hipMalloc((void**)&objects, cell_size);
	hipMalloc((void**)&objects_temp, cell_size);
	hipMalloc((void**)&indices, cell_size);
	hipMalloc((void**)&indices_num, sizeof(uint32_t));
	hipMalloc((void**)&radix_sums, num_radices * sizeof(uint32_t));

	// initialize cells and objects
	init <<< num_blocks, threads_per_block, threads_per_block * sizeof(unsigned int) >>> (cells, objects, balls, length, width, height, gridSize, grid_x, grid_y, grid_z, n);

	// in-place排序，将H cell排在P cell前面
	radixSort(cells, objects, cells_temp, objects_temp, radix_sums, 8 * n, indices, indices_num, num_blocks, threads_per_block);

	uint32_t indices_num_;
	hipMemcpy((void*)&indices_num_, (void*)indices_num, sizeof(uint32_t), hipMemcpyDeviceToHost);

	unsigned int threads_total = num_blocks * threads_per_block;
	unsigned int group_per_thread = indices_num_ / threads_total + 1;
	collision <<< num_blocks, threads_per_block >>> (cells, objects, balls, n, 8 * n, indices, indices_num_, group_per_thread, length, width, height, gridSize, grid_x, grid_y, grid_z);

	hipFree(cells);
	hipFree(cells_temp);
	hipFree(objects);
	hipFree(objects_temp);
	hipFree(indices);
	hipFree(indices_num);
	hipFree(radix_sums);
}

// 基数排序
void radixSort(uint32_t* cells, uint32_t* objects, uint32_t* cells_temp, uint32_t* objects_temp,
	uint32_t* radix_sums, int n, uint32_t* indices, uint32_t* num_indices, unsigned int num_blocks, 
	unsigned int threads_per_block)
{
	for (int i = 0; i < 32; i += RADIX_LENGTH)
	{
		getRadixSum <<< num_blocks, threads_per_block >>> (cells, radix_sums, n, i);

		arrange <<< num_blocks, threads_per_block >>> (cells, objects, cells_temp, objects_temp, radix_sums, n, i);
		
		uint32_t* cells_s = cells;
		cells = cells_temp;
		cells_temp = cells_s;

		uint32_t* objects_s = objects;
		objects = objects_temp;
		objects_temp = objects_s;
	}

	getCellIndex <<< num_blocks, threads_per_block >>> (cells, n, indices, num_indices);
}